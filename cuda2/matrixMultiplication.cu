#include "hip/hip_runtime.h"
// Matrix multiplication by parts
// Elements stored in row-major order

using namespace std;

#include <stdio.h>
#include <iostream>
#include <fstream>
#include "helper_timer.h"

typedef struct {
    int width;
    int height;
    float *elements;
} Matrix;

// Forward declaration of matrix mult
__global__ void MatMulKernel(const Matrix, const Matrix, Matrix);

StopWatchInterface *startTimer();

float stopTimer(StopWatchInterface *timer);

// Host code
void MatMul(const Matrix A, const Matrix B, Matrix C, const int tries, int block_size) {
    // Load matrices A and B to device memory
    Matrix d_A;
    d_A.width = A.width;
    d_A.height = A.height;
    size_t size = A.width * A.height * sizeof(float);
    hipMalloc((void **) &d_A.elements, size);
    hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);

    Matrix d_B;
    d_B.width = B.width;
    d_B.height = B.height;
    size = B.width * B.height * sizeof(float);
    hipMalloc((void **) &d_B.elements, size);
    hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);

    // allocate C in device
    Matrix d_C;
    d_C.width = C.width;
    d_C.height = C.height;
    size = d_C.width * d_C.height * sizeof(float);
    hipMalloc((void **) &d_C.elements, size);

    // call kernel
    dim3 dimBlock(block_size, block_size, 1); // threads per block?
    dim3 dimGrid((B.width - 1) / block_size + 1, (A.height - 1) / block_size + 1); // number of blocks?


    StopWatchInterface *timer = startTimer();

    for (int trie = 0; trie < tries; trie++) {
        MatMulKernel << < dimGrid, dimBlock >> > (d_A, d_B, d_C);
    }
    float time = stopTimer(timer) / tries;

    //size; width; height; block_size; tries; time
    printf("%d; %d; %d; %d; %d; %f\n", C.width * C.height, C.width, C.height, block_size, tries, time);
    // copy C to host
    hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);

    // free device memory
    hipFree(d_A.elements);
    hipFree(d_B.elements);
    hipFree(d_C.elements);
}

float stopTimer(StopWatchInterface *timer) {
    hipDeviceSynchronize();
    sdkStopTimer(&timer);
    float time = sdkGetTimerValue(&timer);
    sdkDeleteTimer(&timer);
    return time;
}

StopWatchInterface *startTimer() {
    StopWatchInterface *timer = NULL;
    sdkCreateTimer(&timer);
    sdkResetTimer(&timer);
    sdkStartTimer(&timer);
    return timer;
}

//matrix multiplication kernel
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C) {
    // each thread computes one element of C and acumulates results to
    float Cvalue = 0;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if ((row >= A.height) || (col >= B.width)) {
        return;
    }
    for (int e = 0; e < A.width; e++) {
        Cvalue += A.elements[row * A.width + e] *
                  B.elements[e * B.width + col];
    }
    C.elements[row * C.width + col] = Cvalue;

}

int main(int argc, char *const argv[]) {
    if (argc < 7) {
        printf("usage: %s <matrixA> <matrixB> <outputC> <Width> <block_size> <tires> \n", argv[0]);
        exit(0);
    }
    char *A_filename = argv[1];
    char *B_filename = argv[2];
    char *C_filename = argv[3];
    int Width = atoi(argv[4]);
    int block_size = atoi(argv[5]);
    int tries = atoi(argv[6]);


    Matrix A;
    Matrix B;
    Matrix C;

    A.width = Width;
    B.width = Width;
    C.width = Width;

    A.height = Width;
    B.height = Width;
    C.height = Width;

    A.elements = new float[Width * Width];
    B.elements = new float[Width * Width];
    C.elements = new float[Width * Width];

    //fill matrices
    std::ifstream A_input;
    std::ifstream B_input;
    A_input.open(A_filename);
    B_input.open(B_filename);

    float a, b;
    A_input >> a;
    B_input >> b;
    int i = 0;
    while (!A_input.eof()) {
        A.elements[i] = a;
        B.elements[i] = b;
        A_input >> a;
        B_input >> b;
        i += 1;
    }
    A_input.close();
    B_input.close();

    MatMul(A, B, C, tries, block_size);
    std::ofstream C_output;
    C_output.open(C_filename);
    for (int i = 0; i < Width; i++) {
        for (int j = 0; j < Width; j++)
            C_output << C.elements[i * Width + j] << "\t";
        C_output << endl;
    }

}
	
