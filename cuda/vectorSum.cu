#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include "helper_timer.h"

__global__ void add (int *a, int *b, int *c, int N) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if(tid < N) {
		c[tid] = a[tid]+b[tid];
	}
}
void add_with_cpu(int *a, int *b, int *c, int N){
	for (int i=0; i < N; i++){
		c[i] = a[i] + b[i];
	}
}

float count ( int , int, int * );

int main( int argc, char **argv )  {
	if (argc < 2){
		printf("usage: %s <vector_size>", argv[0]);
		exit(-1);
	}
	int N = atoi(argv[1]);
	/*if (!strcmp(argv[2], "gpu"))
		use_gpu = 1;
	else if (strcmp(argv[2], "cpu")){
                printf("usage: %s <vector_size> <mode=[gpu|cpu]>\t\t (mode %s is incorrect)", argv[0], argv[2]);
		exit(-1);
	}*/
        int *c = (int *) malloc(N * sizeof(int));
        int *ccpu = (int *) malloc(N * sizeof(int));

	float time = count(N, 1, c);
	printf ("Time for the gpu: %f ms\n", time);

	time = count(N, 1, ccpu);
	printf ("Time for the cpu: %f ms\n", time);

}
float count ( int N, int use_gpu, int * c){


	int *a = (int *) malloc(N * sizeof(int));
	int *b = (int *) malloc(N * sizeof(int));
//	int *c = (int *) malloc(N * sizeof(int));
	int *dev_a, *dev_b, *dev_c;

	if (use_gpu){
		hipMalloc((void**)&dev_a,N * sizeof(int));
		hipMalloc((void**)&dev_b,N * sizeof(int));
		hipMalloc((void**)&dev_c,N * sizeof(int));
	}

	for (int i=0; i<N; i++) {
		a[i] = i;
		b[i] = i*2;
	}
	hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_c, c, N*sizeof(int), hipMemcpyHostToDevice);

	StopWatchInterface *timer=NULL;
	sdkCreateTimer(&timer);
	sdkResetTimer(&timer);
	sdkStartTimer(&timer);
	
	if (use_gpu)
		add <<<1,N>>> (dev_a,dev_b,dev_c, N);
	else 
		add_with_cpu (a, b, c, N);

	hipDeviceSynchronize();
	sdkStopTimer(&timer);
	float time = sdkGetTimerValue(&timer);
	sdkDeleteTimer(&timer);
//	printf ("Time for the kernel: %f ms\n", time);	

	if(use_gpu)
		hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);
	
	for (int i = 0; i < N; i++) {
		printf("%d+%d=%d\n", a[i], b[i], c[i]);
	}

	if (use_gpu){
		hipFree(dev_a);
		hipFree(dev_b);
		hipFree(dev_c);
	}
	free(a);
	free(b);

	return time;
}
