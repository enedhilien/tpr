#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include "helper_timer.h"

__global__ void add (int *a, int *b, int *c) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if(tid < N) {
		c[tid] = a[tid]+b[tid];
	}
}
int main( int arc, char **argv )  {
	if (argc < 2){
		printf("usage: %s <vector_size>", argv[0]);
		exit(-1);
	}
	int N = atoi(argv[1]);

	int *a = malloc(N * sizeof(int));
	int *b = malloc(N * sizeof(int));
	int *c = malloc(N * sizeof(int));
	int *dev_a, *dev_b, *dev_c;

	hipMalloc((void**)&dev_a,N * sizeof(int));
	hipMalloc((void**)&dev_b,N * sizeof(int));
	hipMalloc((void**)&dev_c,N * sizeof(int));
	for (int i=0; i<N; i++) {
		a[i] = i;
		b[i] = i*2;
	}
	hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_c, c, N*sizeof(int), hipMemcpyHostToDevice);

	StopWatchInterface *timer=NULL;
	sdkCreateTimer(&timer);
	sdkResetTimer(&timer);
	sdkStartTimer(&timer);

	add <<<1,N>>> (dev_a,dev_b,dev_c);

	hipDeviceSynchronize();
	sdkStopTimer(&timer);
	float time = sdkGetTimerValue(&timer);
	sdkDeleteTimer(&timer);
	printf ("Time for the kernel: %f ms\n", time);	

	hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);
	for (int i = 0; i < N; i++) {
		printf("%d+%d=%d\n", a[i], b[i], c[i]);
	}
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	return 0;
}
