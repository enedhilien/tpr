#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include "helper_timer.h"

__global__ void add (int *a, int *b, int *c, int N) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if(tid < N) {
		c[tid] = a[tid]+b[tid];
	}
}
void add_with_cpu(int *a, int *b, int *c, int N){
	for (int i=0; i < N; i++){
		c[i] = a[i] + b[i];
	}
}

float count ( int , int, int * , int, int);

int main( int argc, char **argv )  {
	if (argc < 4){
		printf("usage: %s <vector_size> <blocksize> <tries>\n", argv[0]);
		exit(-1);
	}
	int N = atoi(argv[1]);
	int blocksize = atoi(argv[2]);
	int tries = atoi(argv[3]);
	int nblocks = (N - 1) / blocksize + 1;

	/*if (!strcmp(argv[2], "gpu"))
		use_gpu = 1;
	else if (strcmp(argv[2], "cpu")){
                printf("usage: %s <vector_size> <mode=[gpu|cpu]>\t\t (mode %s is incorrect)", argv[0], argv[2]);
		exit(-1);
	}*/
        int *c = (int *) malloc(N * sizeof(int));
        int *ccpu = (int *) malloc(N * sizeof(int));

	printf("%d; %d; %d; ", N, nblocks, blocksize);

	float time = 0;
	for (int trie = 0; trie< tries; trie++){
		time += count(N, 1, c, nblocks, blocksize);
	}
	time = time / tries;

	printf ("%f; ", time);

	time = count(N, 0, ccpu, nblocks, blocksize);
	printf ("%f; ", time);
	
	int i;
	for (i=0; i< N; i++){
		if (ccpu[i] != c[i]){
			break;
		}
		
	}
	printf("%d\n", N - i);
	

}
float count ( int N, int use_gpu, int * c, int nblocks, int blocksize){


	int *a = (int *) malloc(N * sizeof(int));
	int *b = (int *) malloc(N * sizeof(int));
//	int *c = (int *) malloc(N * sizeof(int));
	int *dev_a, *dev_b, *dev_c;

	if (use_gpu){
		hipMalloc((void**)&dev_a,N * sizeof(int));
		hipMalloc((void**)&dev_b,N * sizeof(int));
		hipMalloc((void**)&dev_c,N * sizeof(int));
	}

	for (int i=0; i<N; i++) {
		a[i] = i;
		b[i] = i*2;
	}
	hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_c, c, N*sizeof(int), hipMemcpyHostToDevice);

	StopWatchInterface *timer=NULL;
	sdkCreateTimer(&timer);
	sdkResetTimer(&timer);
	sdkStartTimer(&timer);
	

	if (use_gpu)
		add <<<nblocks,blocksize>>> (dev_a,dev_b,dev_c, N);
	else 
		add_with_cpu (a, b, c, N);

	hipDeviceSynchronize();
	sdkStopTimer(&timer);
	float time = sdkGetTimerValue(&timer);
	sdkDeleteTimer(&timer);
//	printf ("Time for the kernel: %f ms\n", time);	

	if(use_gpu)
		hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);
	
/*	for (int i = 0; i < N; i++) {
		printf("%d+%d=%d\n", a[i], b[i], c[i]);
	}
*/
	if (use_gpu){
		hipFree(dev_a);
		hipFree(dev_b);
		hipFree(dev_c);
	}
	free(a);
	free(b);

	return time;
}
